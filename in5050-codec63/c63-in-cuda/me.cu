#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "me.h"
#include "tables.h"

extern struct c63_common *d_cm;
extern struct frame *d_refframe, *d_curframe;

extern yuv_t *d_curframe_orig, *d_refframe_recons, *d_curframe_predicted;
extern yuv_buf *d_curframe_origbuf, *d_refframe_reconsbuf, *d_curframe_predictedbuf;

extern struct macroblock *d_curframe_mby, *d_curframe_mbu, *d_curframe_mbv;

#define SHUFFLE_FULL_MASK 0xffffffff

__device__ static void sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result)
{
  __shared__ int part_sad_sums[2];

  int lane_index = threadIdx.x
  int warp_index = threadIdx.y;
  int tid = threadIdx.x + threadIdx.y * blockDim.x;

  __syncwarp();

  int shuffled_result = abs(block2[warp_index*stride+lane_index] - block1[warp_index*stride+lane_index])

  for (int offset = 16; offset > 0; offset /= 2) {
    shuffled_result += __shfl_down_sync(SHUFFLE_FULL_MASK, shuffled_result, offset);
  }

  __syncwarp();
  if (lane_index == 0) {
    part_sad_sums[warp_index] = shuffled_result;
  }

  __syncthreads();
  if (tid == 0){
    *result = part_sad_sums[0] + part_sad_sums[1];
  }
}

/* Motion estimation for 8x8 block */
__device__ static void me_block_8x8(
  struct c63_common *cm, 
  int mb_x, 
  int mb_y, 
  uint8_t *orig, 
  uint8_t *ref, 
  int color_component
)
{
  int tid = threadIdx.x + threadIdx.y * blockDim.x;

  struct macroblock *mb = &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  int range = cm->me_search_range;
  
  /* Quarter resolution for chroma channels. */
  if (color_component > 0) { range /= 2; }
  
  int left = mb_x * 8 - range;
  int top = mb_y * 8 - range;
  int right = mb_x * 8 + range;
  int bottom = mb_y * 8 + range;
  
  int w = cm->padw[color_component];
  int h = cm->padh[color_component];

  if (left < 0) { left = 0; }
  if (top < 0) { top = 0; }
  if (right > (w - 8)) { right = w - 8; }
  if (bottom > (h - 8)) { bottom = h - 8; }
  
  int x, y;
  
  int mx = mb_x * 8;
  int my = mb_y * 8;
  
  int best_sad = INT_MAX;
  
  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      int sad;
      sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, &sad);

      if (tid == 0) {
        if (sad < best_sad)
        {
          mb->mv_x = x - mx;
          mb->mv_y = y - my;
          best_sad = sad;
        }
      }
    }
  }
  
  if (tid == 0) {
    mb->use_mv = 1;
  }
}

__device__ void c63_motion_estimate_gpu(struct c63_common *cm) {
  // /* Compare this frame with previous reconstructed frame */
  int color_component = gridDim.z;
  int mb_x = gridDim.x;
  int mb_y = gridDim.y;

  if (color_component == 0) {
    // Y component
    me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->Y, cm->refframe->recons->Y, Y_COMPONENT);
  } else if(color_component == 1) {
    // U component
    if (mb_x < cm->uv_mb_cols && mb_y < cm->uv_mb_rows) {
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->U, cm->refframe->recons->U, U_COMPONENT); 
    }
  } else {
    // V component
    if (mb_x < cm->uv_mb_cols && mb_y < cm->uv_mb_rows) {
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->V, cm->refframe->recons->V, V_COMPONENT); 
    }
  }
}

/* Motion compensation for 8x8 block */
__device__ static void mc_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
  uint8_t *predicted, uint8_t *ref, int color_component)
{
  struct macroblock *mb = &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  if (!mb->use_mv) return;
  
  int w = cm->padw[color_component];

  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  
  int y = (mb_y * 8) + (tid >> 3);
  int x = (mb_x * 8) + (tid & 0x7);
  
  /* Bank conflicts? Bank conflicts may not be a problem for global memory */
  predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
}

__device__ void c63_motion_compensate_gpu(struct c63_common *cm) {
  int color_component = gridDim.z;
  int mb_x = gridDim.x;
  int mb_y = gridDim.y;

  __syncthreads();

  if (color_component == 0) {
    // Y component
    mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y, cm->refframe->recons->Y, Y_COMPONENT);
  } else if(color_component == 1) {
    // U component
    if (mb_x < cm->uv_mb_cols && mb_y < cm->uv_mb_rows) {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U, cm->refframe->recons->U, U_COMPONENT);
    }
  } else {
    // V component
    if (mb_x < cm->uv_mb_cols && mb_y < cm->uv_mb_rows) {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V, cm->refframe->recons->V, V_COMPONENT);
    }
  }
}

__global__ void c63_estimate_compensate_gpu(struct c63_common *cm) {
  c63_motion_estimate_gpu(cm);
  c63_motion_compensate_gpu(cm);
}

void c63_estimate_compensate(struct c63_common *cm) {
  /* Copy into device curframe original buffer */
  hipMemcpy(
    d_curframe_origbuf, 
    cm->curframe->orig->buf,
    cm->total_yuv_buflen,
    hipMemcpyHostToDevice
  );

  /* Copy into device refframe recons buffer */
  hipMemcpy(
    d_refframe_reconsbuf, 
    cm->refframe->recons->buf,
    cm->total_yuv_buflen,
    hipMemcpyHostToDevice
  );


  dim3 grid(cm->mb_cols, cm->mb_rows, 3);
  dim3 blk(32, 2, 1);

  c63_estimate_compensate <<< grid, blk >>>(d_cm);

  hipDeviceSynchronize();

  /* Copy back curframe predicted buffer to the host */
  hipMemcpy(
    cm->curframe->predicted->buf,
    d_curframe_predictedbuf,
    cm->total_yuv_buflen,  
    hipMemcpyDeviceToHost
  );

  /* Copy back current frame macroblocks (Y, U and V) */
  hipMemcpy(
    cm->curframe->mbs[Y_COMPONENT],
    d_curframe_mby,
    cm->y_mb_buflen, 
    hipMemcpyDeviceToHost
  );
  
  hipMemcpy(
    cm->curframe->mbs[U_COMPONENT],
    d_curframe_mbu, 
    cm->u_mb_buflen, 
    hipMemcpyDeviceToHost
  );
  
  hipMemcpy(
    cm->curframe->mbs[V_COMPONENT],
    d_curframe_mbv,
    cm->v_mb_buflen, 
    hipMemcpyDeviceToHost
  );
}