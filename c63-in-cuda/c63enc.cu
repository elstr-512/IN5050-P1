#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stddef.h>

#include "c63.h"
#include "c63_write.h"
#include "quantdct.h"
#include "common.h"
#include "me.h"
#include "tables.h"

static char *output_file, *input_file;
FILE *outfile;

static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;

/* Created pointer globals for VRAM memory */

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static yuv_t* read_yuv(FILE *file, struct c63_common *cm)
{
  size_t len = 0;
  yuv_t *image = (yuv_t*)malloc(sizeof(yuv_t));
  yuv_buf img_yuv_buf;

  /* Allocating space for Y, U and v pixel components */
  img_yuv_buf = (yuv_buf)calloc(cm->total_yuv_buflen, 1);

  /* Read Y, U and V. U and V components are sampled 4:2:0 meaning 1/4 size of Y */
  len += fread(img_yuv_buf               , 1, cm->y_datalen, file);
  len += fread(img_yuv_buf + cm->u_bufoff, 1, cm->u_datalen, file);
  len += fread(img_yuv_buf + cm->v_bufoff, 1, cm->v_datalen, file);

  if (ferror(file))
  {
    perror("ferror");
    exit(EXIT_FAILURE);
  }

  if (feof(file))
  {
    free(img_yuv_buf);
    free(image);
    return NULL;
  }
  else if (len != width*height*1.5)
  {
    fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
    fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);
    
    free(img_yuv_buf);
    free(image);
    return NULL;
  }

  image->Y   = img_yuv_buf;
  image->U   = img_yuv_buf + cm->u_bufoff;
  image->V   = img_yuv_buf + cm->v_bufoff;
  image->buf = img_yuv_buf;

  return image;
}

static void c63_encode_image(struct c63_common *cm, yuv_t *image)
{  
  /* Advance to next frame */
  destroy_frame(cm->refframe);
  cm->refframe = cm->curframe;
  cm->curframe = create_frame(cm, image);

  /* Check if keyframe */
  if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
  {
    cm->curframe->keyframe = 1;
    cm->frames_since_keyframe = 0;

    fprintf(stderr, " (keyframe) ");
  }
  else { cm->curframe->keyframe = 0; }

  if (!cm->curframe->keyframe)
  {
    c63_estimate_compensate(cm);
  }

  /* DCT and Quantization */
  dct_quantize(image->Y, cm->curframe->predicted->Y, cm->padw[Y_COMPONENT],
      cm->padh[Y_COMPONENT], cm->curframe->residuals->Ydct,
      cm->quanttbl[Y_COMPONENT]);

  dct_quantize(image->U, cm->curframe->predicted->U, cm->padw[U_COMPONENT],
      cm->padh[U_COMPONENT], cm->curframe->residuals->Udct,
      cm->quanttbl[U_COMPONENT]);

  dct_quantize(image->V, cm->curframe->predicted->V, cm->padw[V_COMPONENT],
      cm->padh[V_COMPONENT], cm->curframe->residuals->Vdct,
      cm->quanttbl[V_COMPONENT]);

  /* Reconstruct frame for inter-prediction */
  dequantize_idct(cm->curframe->residuals->Ydct, cm->curframe->predicted->Y,
      cm->ypw, cm->yph, cm->curframe->recons->Y, cm->quanttbl[Y_COMPONENT]);
  dequantize_idct(cm->curframe->residuals->Udct, cm->curframe->predicted->U,
      cm->upw, cm->uph, cm->curframe->recons->U, cm->quanttbl[U_COMPONENT]);
  dequantize_idct(cm->curframe->residuals->Vdct, cm->curframe->predicted->V,
      cm->vpw, cm->vph, cm->curframe->recons->V, cm->quanttbl[V_COMPONENT]);

  /* Function dump_image(), found in common.c, can be used here to check if the
     prediction is correct */

  write_frame(cm);

  ++cm->framenum;
  ++cm->frames_since_keyframe;
}

struct c63_common* init_c63_enc(int width, int height)
{
  int i;

  /* calloc() sets allocated memory to zero */
  c63_common *cm = (c63_common*)calloc(1, sizeof(struct c63_common));

  cm->width = width;
  cm->height = height;

  cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t)(ceil(width/16.0f)*16);
  cm->padh[Y_COMPONENT] = cm->yph = (uint32_t)(ceil(height/16.0f)*16);
  cm->padw[U_COMPONENT] = cm->upw = (uint32_t)(ceil(width*UX/(YX*8.0f))*8);
  cm->padh[U_COMPONENT] = cm->uph = (uint32_t)(ceil(height*UY/(YY*8.0f))*8);
  cm->padw[V_COMPONENT] = cm->vpw = (uint32_t)(ceil(width*VX/(YX*8.0f))*8);
  cm->padh[V_COMPONENT] = cm->vph = (uint32_t)(ceil(height*VY/(YY*8.0f))*8);

  cm->y_buflen = cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT];
  cm->u_buflen = cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT];
  cm->v_buflen = cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT];

  cm->total_yuv_buflen = cm->y_buflen + cm->u_buflen + cm->v_buflen;

  cm->u_bufoff = cm->y_buflen;
  cm->v_bufoff = cm->y_buflen + cm->u_buflen;

  cm->y_datalen = width * height;
  cm->u_datalen = cm->y_datalen / 4;
  cm->v_datalen = cm->u_datalen;

  cm->mb_cols = cm->ypw / 8;
  cm->mb_rows = cm->yph / 8;

  cm->uv_mb_cols = cm->mb_cols / 2;
  cm->uv_mb_rows = cm->mb_rows / 2;

  cm->y_mb_buflen = cm->mb_rows * cm->mb_cols * sizeof(struct macroblock);
  cm->u_mb_buflen = cm->y_mb_buflen / 4;
  cm->v_mb_buflen = cm->u_mb_buflen;

  /* Quality parameters -- Home exam deliveries should have original values,
   i.e., quantization factor should be 25, search range should be 16, and the
   keyframe interval should be 100. */
  cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
  cm->me_search_range = 16;     // Pixels in every direction
  cm->keyframe_interval = 100;  // Distance between keyframes

  /* Initialize quantization tables */
  for (i = 0; i < 64; ++i)
  {
    cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
  }

  return cm;
}

void free_c63_enc(struct c63_common* cm)
{
  destroy_frame(cm->curframe);
  free(cm);
}

struct c63_common *d_cm;
struct frame *d_refframe, *d_curframe;

yuv_t *d_curframe_orig, *d_refframe_recons, // Computing from this (copied from host per frame)
  *d_curframe_predicted; // Computing into this and copying back
yuv_buf d_curframe_origbuf, d_refframe_reconsbuf,
  d_curframe_predictedbuf;

struct macroblock *d_curframe_mby, *d_curframe_mbu, *d_curframe_mbv;

void init_device_state(struct c63_common *cm) {
  /* Allocating necessary memory on the device */
  cudaMallocErr((void**)&d_cm, sizeof(struct c63_common));

  cudaMallocErr((void**)&d_refframe, sizeof(struct frame));
  cudaMallocErr((void**)&d_curframe, sizeof(struct frame));

  cudaMallocErr((void**)&d_curframe_orig, sizeof(struct yuv));
  cudaMallocErr((void**)&d_refframe_recons, sizeof(struct yuv));
  cudaMallocErr((void**)&d_curframe_predicted, sizeof(struct yuv));

  cudaMallocErr((void**)&d_curframe_origbuf, cm->total_yuv_buflen);
  cudaMallocErr((void**)&d_refframe_reconsbuf, cm->total_yuv_buflen);
  cudaMallocErr((void**)&d_curframe_predictedbuf, cm->total_yuv_buflen);

  cudaMallocErr((void**)&d_curframe_mby, cm->y_mb_buflen);
  cudaMallocErr((void**)&d_curframe_mbu, cm->u_mb_buflen);
  cudaMallocErr((void**)&d_curframe_mbv, cm->v_mb_buflen);

  /* Initializing device memory structures */
  cudaMemcpyErr(d_cm, cm, sizeof(struct c63_common), hipMemcpyHostToDevice);

  cudaMemcpyErr((uint8_t*)d_cm + offsetof(struct c63_common, refframe), &d_refframe, sizeof(d_refframe), hipMemcpyHostToDevice);
  cudaMemcpyErr((uint8_t*)d_cm + offsetof(struct c63_common, curframe), &d_curframe, sizeof(d_curframe), hipMemcpyHostToDevice);

  cudaMemcpyErr((uint8_t*)d_curframe + offsetof(struct frame, orig), &d_curframe_orig, sizeof(d_curframe_orig), hipMemcpyHostToDevice);
  cudaMemcpyErr((uint8_t*)d_refframe + offsetof(struct frame, recons), &d_refframe_recons, sizeof(d_refframe_recons), hipMemcpyHostToDevice);
  cudaMemcpyErr((uint8_t*)d_curframe + offsetof(struct frame, predicted), &d_curframe_predicted, sizeof(d_curframe_predicted), hipMemcpyHostToDevice);  

  struct yuv curframe_orig_yuv = {
    .Y   = (uint8_t*)d_curframe_origbuf,
    .U   = (uint8_t*)d_curframe_origbuf + cm->u_bufoff,
    .V   = (uint8_t*)d_curframe_origbuf + cm->v_bufoff,
    .buf = d_curframe_origbuf
  };

  struct yuv refframe_recons_yuv = {
    .Y   = (uint8_t*)d_refframe_reconsbuf,
    .U   = (uint8_t*)d_refframe_reconsbuf + cm->u_bufoff,
    .V   = (uint8_t*)d_refframe_reconsbuf + cm->v_bufoff,
    .buf = d_refframe_reconsbuf
  };
  
  struct yuv curframe_predicted_yuv = {
    .Y   = (uint8_t*)d_curframe_predictedbuf,
    .U   = (uint8_t*)d_curframe_predictedbuf + cm->u_bufoff,
    .V   = (uint8_t*)d_curframe_predictedbuf + cm->v_bufoff,
    .buf = d_curframe_predictedbuf
  };

  cudaMemcpyErr(d_curframe_orig, &curframe_orig_yuv, sizeof(struct yuv), hipMemcpyHostToDevice);
  cudaMemcpyErr(d_refframe_recons, &refframe_recons_yuv, sizeof(struct yuv), hipMemcpyHostToDevice);
  cudaMemcpyErr(d_curframe_predicted, &curframe_predicted_yuv, sizeof(struct yuv), hipMemcpyHostToDevice);

  cudaMemcpyErr(
    (uint8_t*)d_curframe + offsetof(struct frame, mbs), 
    &d_curframe_mby, sizeof(d_curframe_mby), 
    hipMemcpyHostToDevice
  );
  
  cudaMemcpyErr(
    (uint8_t*)d_curframe + offsetof(struct frame, mbs) + sizeof(struct macroblock*), 
    &d_curframe_mbu, sizeof(d_curframe_mbu), 
    hipMemcpyHostToDevice
  );
  
  cudaMemcpyErr(
    (uint8_t*)d_curframe + offsetof(struct frame, mbs) + 2 * sizeof(struct macroblock*), 
    &d_curframe_mbv, sizeof(d_curframe_mbv), 
    hipMemcpyHostToDevice
  );
}

void fini_device_state() {
  cudaFreeErr(d_cm);
  cudaFreeErr(d_refframe);
  cudaFreeErr(d_curframe);
  cudaFreeErr(d_curframe_orig);
  cudaFreeErr(d_refframe_recons);
  cudaFreeErr(d_curframe_predicted);
  cudaFreeErr(d_curframe_origbuf);
  cudaFreeErr(d_refframe_reconsbuf);
  cudaFreeErr(d_curframe_predictedbuf);
  cudaFreeErr(d_curframe_mby);
  cudaFreeErr(d_curframe_mbu);
  cudaFreeErr(d_curframe_mbv);
}

static void print_help()
{
  printf("Usage: ./c63enc [options] input_file\n");
  printf("Commandline options:\n");
  printf("  -h                             Height of images to compress\n");
  printf("  -w                             Width of images to compress\n");
  printf("  -o                             Output file (.c63)\n");
  printf("  [-f]                           Limit number of frames to encode\n");
  printf("\n");

  exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
  int c;
  yuv_t *image;

  if (argc == 1) { print_help(); }

  while ((c = getopt(argc, argv, "h:w:o:f:i:")) != -1)
  {
    switch (c)
    {
      case 'h':
        height = atoi(optarg);
        break;
      case 'w':
        width = atoi(optarg);
        break;
      case 'o':
        output_file = optarg;
        break;
      case 'f':
        limit_numframes = atoi(optarg);
        break;
      default:
        print_help();
        break;
    }
  }

  if (optind >= argc)
  {
    fprintf(stderr, "Error getting program options, try --help.\n");
    exit(EXIT_FAILURE);
  }

  /* Opening file descriptor for input file */
  input_file = argv[optind];
  FILE *infile = fopen(input_file, "rb");
  if (infile == NULL)
  {
    perror("fopen");
    exit(EXIT_FAILURE);
  }

  /* Opening file descriptor for output file */
  outfile = fopen(output_file, "wb");
  if (outfile == NULL)
  {
    perror("fopen");
    exit(EXIT_FAILURE);
  }

  /* Prompting information about limiting the # of encoded frames */
  if (limit_numframes) { printf("Limited to %d frames.\n", limit_numframes); }

  /* Initializing information about the frames that will be encoded */
  struct c63_common *cm = init_c63_enc(width, height);
  cm->e_ctx.fp = outfile;

  /* Initialize device state */
  init_device_state(cm);

  /* Encode input frames */
  int numframes = 0;

  while (1)
  {
    image = read_yuv(infile, cm);

    if (!image) { break; }

    printf("Encoding frame %d, ", numframes);
    c63_encode_image(cm, image);

    free(image->buf);
    free(image);

    printf("Done!\n");

    ++numframes;

    if (limit_numframes && numframes >= limit_numframes) { break; }
  }

  fini_device_state();
  free_c63_enc(cm);
  fclose(outfile);
  fclose(infile);

  return EXIT_SUCCESS;
}
